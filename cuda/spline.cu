#include "hip/hip_runtime.h"
//
// Created by davide miro on 24/09/24.
//

#include <iostream>
#include <cmath>
#include <torch/extension.h>
#include <stdio.h>
#define DIMS batch_size, num_input, num_knots, degree

using namespace std;


//2d tensor idx computation
__device__ size_t compute_idx(int dim, int i, int j){
    return i * dim + j;
}


//compute idx b_spline_basis
__device__ size_t compute_idx_base(int z, int i, int j, int d,
                                      int batch_size, int num_input, int num_knots, int degree){

    int stride_degree = num_input * num_knots * batch_size;
    int stride_batch_size = num_input * num_knots;
    int stride_num_input = num_knots;



    return (d * stride_degree) + (z * stride_batch_size) + (i * stride_num_input) + j

}

__global__ void b_spline_base(float* b_spline_basis, float* x, int batch_size, int num_input, int num_knots, int degree, float* knots) {
    /*
     * z : z-th batch element
     * i : i-th element of the input
     * j : j-th knot
     * k : degree
     */

    int z = blockIdx.x;
    int i = threadIdx.x;

    //dynamic cache
    extern __shared__ float cache_ptr[];
    float* knots_cache = cache_ptr;

    //coalesce load to cache, using grid-stride loop to handle the case batch_size * num_input < num_knots
    for (int x = blockIdx.x * blockDim.x + threadIdx.x;
         x < num_knots;
         x += blockDim.x * gridDim.x){
        knots_cache[x] = knots[x];
    }
    __syncthreads();




    float t;
    float leftTerm = 0.0;
    float rightTerm = 0.0;
    size_t idx = 0;
    size_t idx_ = 0;

    if(z >= batch_size || i >= num_input){
        return;
    }


    for(int d = 0; d <= degree; d++) {
        for (int j = 0; j < num_knots; j++) {

            idx = compute_idx_base(z, i, j, d, DIMS);
            t = x[compute_idx(num_input, z, i)];
            if (d == 0) {
                if (knots[compute_idx(num_knots, i, j)]<= t && t < knots[compute_idx(num_knots, i, j + 1)]) {
                    b_spline_basis[idx] = 1.0;
                } else {
                    b_spline_basis[idx] = 0.0;
                }
            } else {

                if (knots[compute_idx(num_knots, i, j + d)] != knots[compute_idx(num_knots, i, j)]) {
                    idx_ = compute_idx_base(z, i, j, d - 1, DIMS);
                    leftTerm = (t - knots[compute_idx(num_knots,i,j)]) / (knots[compute_idx(num_knots, i, j + d)] - knots[compute_idx(num_knots, i, j)] * b_spline_basis[idx_]);
                }

                if (knots[compute_idx(num_knots, i, j + d + 1)] != knots[compute_idx(num_knots, i, j + 1)]) {
                    idx_ = compute_idx_base(z, i, j + 1, d - 1, DIMS);
                    rightTerm = (knots[compute_idx(num_knots, i, j + d + 1)] - t) / (knots[compute_idx(num_knots, i, j + d + 1)] - knots[compute_idx(num_knots, i, j + 1)]) * b_spline_basis[idx_];
                }
                b_spline_basis[idx] = leftTerm + rightTerm;
            }
        }
    }

}


__device__ float spline(float* cps, float* b_spline_basis, int z, int i, int j, int batch_size, int num_input, int num_knots, int degree) {
    /*
     * z : z-th batch element
     * i : i-th element of the input
     * j : j-th activation function
     * k : k-th knot
     * d : degree
     */

    float result = 0.0;
    size_t idx = compute_idx_base(z, i, j, degree, DIMS);

    for(int k = 0; k < num_knots; k++){
        result = result + (cps[compute_idx(num_knots, i, k)] * b_spline_basis[idx]);
    }

    return result;
}


