#include "hip/hip_runtime.h"
//
// Created by davide miro on 07/09/24.
//

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <ATen/ATen.h>


#include "spline.cu"

#define MAX_DIM 1024


using namespace std;


namespace cuda_kan {

    __device__ float silu(float x) {
        return 1 / (1 + expf(x * -1));

    }



    float **tensor_to_float_ptr(at::Tensor x) {
        // Ensure the tensor is of type float and has 2 dimensions (batch_size, length)
        TORCH_CHECK(x.scalar_type() == at::kFloat, "Tensor must be of type float");
        TORCH_CHECK(x.dim() == 2, "Tensor must be 2D");

        // Get dimensions of the tensor
        int64_t batch_size = x.size(0);
        int64_t length = x.size(1);

        // Get a pointer to the raw data
        float *data_ptr = x.data_ptr<float>();

        // Allocate memory for the array of float pointers (for each row)
        float **float_ptr = new float *[batch_size];

        // Fill the float_ptr array, each element points to a row in the tensor
        for (int64_t i = 0; i < batch_size; ++i) {
            float_ptr[i] = data_ptr + i * length;
        }

        return float_ptr;
    }

    __global__ void kan_activation_function(float **x, float **y, const float *wb, const float *ws, const float *cps, const float *knots, const float ***bSplineBasis, int k, int N) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y;
        int z = blockIdx.z;
        float result = 0.0;
        if (i < N) {
            result = wb[j] * silu(x[z][i]) + ws[j] * b_spline(i, cps, knots, bSplineBasis, k);
            atomicAdd(&y[z][i], result);
        }

    }


    at::Tensor kan_layer(at::Tensor x, at::Tensor wb, at::Tensor ws, at::Tensor knots, at::Tensor cps) {

        TORCH_CHECK(wb.size(0) < MAX_DIM); //TODO: review check
        TORCH_CHECK(knots.size(0) < MAX_DIM);
        TORCH_CHECK(cps.size(0) < MAX_DIM);


        TORCH_CHECK(x.dtype() == at::kFloat);
        TORCH_CHECK(wb.dtype() == at::kFloat);
        TORCH_CHECK(ws.dtype() == at::kFloat);
        TORCH_CHECK(wb.dtype() == at::kFloat);
        TORCH_CHECK(ws.dtype() == at::kFloat);

        TORCH_INTERNAL_ASSERT(x.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(wb.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(ws.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(knots.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(cps.device().type() == at::DeviceType::CUDA);


        at::Tensor x_contig = x.contiguous();
        at::Tensor wb_contig = wb.contiguous();
        at::Tensor ws_contig = ws.contiguous();
        at::Tensor cps_contig = cps.contiguous();
        at::Tensor knots_contig = knots.contiguous();

        at::Tensor y = torch::zeros({x.size(0), wb.size(0)}, wb_contig.options());

        float **x_ptr = tensor_to_float_ptr(x_contig);
        const float *wb_ptr = wb_contig.data_ptr<float>();
        const float *ws_ptr = ws_contig.data_ptr<float>();
        const float *cps_ptr = cps_contig.data_ptr<float>();
        const float *knots_ptr = knots_contig.data_ptr<float>();

        float **y_ptr = tensor_to_float_ptr(y);



        //TODO: k as argument of CUDA/CPP function
        int k = 3; //degree
        int num_input = x.size(1);
        int num_activations = wb.size(0);
        int num_threads = 1024; //max number of threads x bloc

        dim3 num_blocks(num_input / 1024, num_activations, x.size(0)); // num_input x num_activations x batch_size

        kan_activation_function<<<num_blocks, num_threads>>>(x_ptr, y_ptr, wb_ptr, ws_ptr, cps_ptr, knots_ptr, k,
                                                             num_activations);


        return y;


    }

    PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}

    TORCH_LIBRARY(cuda_kan, m) {
        m.def("kan_layer(Tensor x, Tensor wb, Tensor ws, Tensor knots, Tensor cps) -> Tensor");
    }

    TORCH_LIBRARY_IMPL(cuda_kan, CUDA, m) {
        m.impl("kan_layer", &kan_layer);
    }

}


