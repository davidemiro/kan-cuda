#include "hip/hip_runtime.h"
//
// Created by davide miro on 07/09/24.
//

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <ATen/ATen.h>


#include "cpp/spline.cpp"

#define MAX_DIM 1024


using namespace std;


namespace cuda_kan {

    float silu(float x) {
        return 1 / (1 + expf(x * -1));

    }

    float **tensor_to_float_ptr(at::Tensor x) {
        // Ensure the tensor is of type float and has 2 dimensions (batch_size, length)
        TORCH_CHECK(x.scalar_type() == at::kFloat, "Tensor must be of type float");
        TORCH_CHECK(x.dim() == 2, "Tensor must be 2D");

        // Get dimensions of the tensor
        int64_t batch_size = x.size(0);
        int64_t length = x.size(1);

        // Get a pointer to the raw data
        float *data_ptr = x.data_ptr<float>();

        // Allocate memory for the array of float pointers (for each row)
        float **float_ptr = new float *[batch_size];

        // Fill the float_ptr array, each element points to a row in the tensor
        for (int64_t i = 0; i < batch_size; ++i) {
            float_ptr[i] = data_ptr + i * length;
        }

        return float_ptr;
    }

    __global__ void kan_activation_function(float *x, float *y, float *wb, float *ws, float *cps, float *knots, int k, int N) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y;
        int z = blockIdx.z;
        if (i < N) {
            y[z][j] = y[z][j] + wb[j] * silu(x[i]) + ws[j] * b_spline(x[z][i], N, cps, knots, k);
        }

    }


    at::Tensor kan_layer(at::Tensor x, at::Tensor wb, at::Tensor ws, at::Tensor knots, at::Tensor cps) {

        TORCH_CHECK(wb.size(0) < MAX_DIM); //TODO: review check
        TORCH_CHECK(knots.size(0) < MAX_DIM);
        TORCH_CHECK(cps.size(0) < MAX_DIM);


        TORCH_CHECK(x.dtype() == at::kFloat);
        TORCH_CHECK(wb.dtype() == at::kFloat);
        TORCH_CHECK(ws.dtype() == at::kFloat);
        TORCH_CHECK(wb.dtype() == at::kFloat);
        TORCH_CHECK(ws.dtype() == at::kFloat);

        TORCH_INTERNAL_ASSERT(x.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(wb.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(ws.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(knots.device().type() == at::DeviceType::CUDA);
        TORCH_INTERNAL_ASSERT(cps.device().type() == at::DeviceType::CUDA);


        at::Tensor x_contig = x.contiguous();
        at::Tensor wb_contig = wb.contiguous();
        at::Tensor ws_contig = ws.contiguous();
        at::Tensor cps_contig = controlPoints.contiguous();
        at::Tensor knots_contig = knots.contiguous();

        at::Tensor y = torch::zeros({x.size(0), wb.size(0)}, wb_contig.options());

        float **x_ptr = tensor_to_float_ptr(x_contig);
        const float *wb_ptr = wb_contig.data_ptr<float>();
        const float *ws_ptr = ws_contig.data_ptr<float>();
        const float *cps_ptr = controlPoints_contig.data_ptr<float>();
        const float *knots_ptr = knots_contig.data_ptr<float>();

        float **y_ptr = tensor_to_float_ptr(y);

        int num_cps = cps.size(0);
        //TODO: k deve essere passato come argomento
        int k = 3

        int num_threads = 1024; //max number of threads x bloc
        dim3 num_blocks(N / 1024, M, x.size(0)) // num_input x num_activations x batch_size

        kan_activation_function<<<num_blocks, num_threads>>>(x_ptr, y_ptr, wb_ptr, ws_ptr, cps_ptr, knots_ptr, k,
                                                             num_cps);


        return y;


    }

    PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}

    TORCH_LIBRARY(cuda_kan, m) {
        m.def("kan_layer(Tensor x, Tensor wb, Tensor ws, Tensor knots, Tensor cps) -> Tensor");
    }

    TORCH_LIBRARY_IMPL(cuda_kan, CUDA, m) {
        m.impl("kan_layer", &kan_layer);
    }

}


